#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA
#include ""

#define pi 3.1415926535
#define LENGTH 1000 //signal sampling points
int main()
{
    // data gen
    float Data[LENGTH] = { 1,2,3,4 };
    float fs = 1000000.000;//sampling frequency
    float f0 = 200000.00;// signal frequency
    for (int i = 0; i < LENGTH; i++)
    {
        Data[i] = 12 * cos(2 * pi * f0 * i / fs);//signal gen,
    }

    hipfftComplex* CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
    int i;
    for (i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex* d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device
    double startTime = clock();
    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host
    double endTime = clock();
    for (i = 0; i < LENGTH / 2; i++)
    {
        printf("i=%d\tf= %6.1fHz\tRealAmp=%3.1f\t", i, fs * i / LENGTH, CompData[i].x * 2.0 / LENGTH);
        printf("ImagAmp=+%3.1fi", CompData[i].y * 2.0 / LENGTH);
        printf("\n");
    }
    printf("Time=%fms\n", (endTime - startTime) / CLOCKS_PER_SEC * 1000);
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

}